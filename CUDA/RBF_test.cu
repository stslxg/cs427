#include <stdio.h>
#include "math.h"
#include "hip/hip_runtime.h"


#define M 500
#define N 1000
#define TILE_WIDTH 16
const float sigma = 100.0f;

float x[M][N];
float k[M][M];

__global__ void test_kernel(float *dK) {
#define IDX_k(i,j) ((i) * M + (j))

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < M && j < M)
    dK[IDX_k(i,j)] = exp(dK[IDX_k(i,j)]);
}

void init() {
  int i,j;
  for(i = 0; i < M; ++i)
    for(j = 0; j < N; ++j)
      x[i][j] = (i + j) / 2000.0f;
}

void work() {
  int i,j,ii;
  for(i = 0; i < M; ++i)
    for(j = 0; j < M; ++j) {
      k[i][j] = 0;
      for (ii = 0; ii < N; ++ii)
	k[i][j] += (x[i][ii] - x[j][ii]) * (x[i][ii] - x[j][ii]);
      k[i][j] = - k[i][j] / (2.0 * sigma * sigma);
    }

  hipSetDevice(0);
  float *dK = NULL;
  hipMalloc((void**) &dK, sizeof(float) * M * M);
  hipMemcpy(dK, k, sizeof(float) * M * M, hipMemcpyHostToDevice);

  dim3 DimGrid((M + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);
  dim3 DimBlock(TILE_WIDTH, TILE_WIDTH);
  
  test_kernel<<<DimGrid, DimBlock>>>(dK);
  
  hipMemcpy(k, dK, sizeof(float) * M * M, hipMemcpyDeviceToHost);
  hipFree(dK);
}

void output() {
  int i,j;
  for(i = 0; i < M; ++i) {
    for (j = 0; j < M; ++j)
      printf("%f ", k[i][j]);
    printf("\n");
  }
}

int main() {
  init();
  work();
  output();
  return 0;
}
