#include <stdio.h>
#include "math.h"
#include "hip/hip_runtime.h"


#define M 5000
#define N 10000
#define TILE_WIDTH 8
const float sigma = 100.0f;

float x[M][N];
float k[M][M];

__global__ void RBFkernel(float *dX, float *dK) {

#define IDX_k(i,j) ((i) * M + (j))
#define IDX(i,j) ((i) * N + (j))

  __shared__ float TILE_i[TILE_WIDTH][TILE_WIDTH];
  __shared__ float TILE_j[TILE_WIDTH][TILE_WIDTH];

  float value = 0;
  float value1 = 0;
  
  int bi = blockIdx.x * blockDim.x;
  int bj = blockIdx.y * blockDim.y;
  int ti = threadIdx.x;
  int tj = threadIdx.y;
  
  int i = bi + ti;
  int j = bj + tj;

  int m,k;
  for (m = 0; m < (N + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
    if (bi + ti < M && m * TILE_WIDTH + tj < N)
      TILE_i[ti][tj] = dX[IDX(bi + ti, m * TILE_WIDTH + tj)];
    if (bj + ti < M && m * TILE_WIDTH + tj < N)
      TILE_j[ti][tj] = dX[IDX(bj + ti, m * TILE_WIDTH + tj)];
    __syncthreads();
    
    if (i < M && j < M)
      for (k = 0; k < TILE_WIDTH; ++k) {
	if (m * TILE_WIDTH + k < N)
	  value += (TILE_i[ti][k] - TILE_j[tj][k]) * (TILE_i[ti][k] - TILE_j[tj][k]);
	/*	if (m * TILE_WIDTH + k + TILE_WIDTH / 2 < N) 
		value1 += (TILE_i[ti][k + TILE_WIDTH / 2] - TILE_j[tj][k + TILE_WIDTH / 2]) * (TILE_i[ti][k + TILE_WIDTH / 2] - TILE_j[tj][k + TILE_WIDTH / 2]); */
      }
    __syncthreads();
  }

  if (i < M && j < M) {
    dK[IDX_k(i,j)] = exp(- (value + value1) / (2 * sigma * sigma));
  }
}

void init() {
  int i,j;
  for(i = 0; i < M; ++i)
    for(j = 0; j < N; ++j)
      x[i][j] = (i + j) / 20000.0f;
}

void calculateRBFkernel() {
  hipSetDevice(0);

  float *dX = NULL;
  float *dK = NULL;
  
  hipMalloc((void**) &dX, sizeof(float) * M * N);
  hipMalloc((void**) &dK, sizeof(float) * M * M);
  hipMemcpy(dX, x, sizeof(float) * M * N, hipMemcpyHostToDevice);

  dim3 DimGrid((M + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);
  dim3 DimBlock(TILE_WIDTH, TILE_WIDTH);

  RBFkernel<<<DimGrid, DimBlock>>>(dX, dK);

  hipMemcpy(k, dK, sizeof(float) * M * M, hipMemcpyDeviceToHost);
  hipFree(dX);
  hipFree(dK);
}

void output() {
  /*int i,j;
      for(i = 0; i < M; ++i) {
    for (j = 0; j < M; ++j)
      printf("%f ", k[i][j]);
    printf("\n");
    }*/
}

int main() {
  init();
  calculateRBFkernel();
  output();
  return 0;
}
